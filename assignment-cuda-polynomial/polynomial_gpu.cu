
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>


__global__ void polynomial_expansion (float* poly, int degree, int n, float* array)
{
    float out = 0.;
    float xdegree = 1.;
  
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < n)
    {
        float x = array[i];
        for (int k=0; k<=degree; ++k)
        {
            out += xdegree*poly[k];
            xdegree *= x;
        }
        array[i] = out;
    }


}


int main (int argc, char* argv[])
{
  
    if (argc < 3)
    {
        std::cerr<<"usage: "<<argv[0]<<" n degree"<<std::endl;
        return -1;
    }

    int n = atoi(argv[1]); 
    int degree = atoi(argv[2]);
    int nbiter = 1;

    float* array = new float[n];
    float* poly = new float[degree+1];
    for (int i=0; i<n; ++i)
        array[i] = 1.;

    for (int i=0; i<degree+1; ++i)
        poly[i] = 1.;

  
    float *xArrayPointer, *coefficientArrayPointer; 

    //Allocating space in device
    hipMalloc(&xArrayPointer, n*sizeof(float)); 
    hipMalloc(&coefficientArrayPointer, (degree+1)*sizeof(float)); 
 
    //copying arrays to device array
    hipMemcpy(xArrayPointer, array, n*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(coefficientArrayPointer, poly, (degree+1)*sizeof(float), hipMemcpyHostToDevice); 
    
    std::chrono::time_point<std::chrono::system_clock> begin, end;
    begin = std::chrono::system_clock::now();
  
  
    for (int iter = 0; iter<nbiter; ++iter)
    {
        polynomial_expansion<<<(n+512)/512+1, 512>>>(coefficientArrayPointer, degree, n, xArrayPointer);
    }

    hipDeviceSynchronize(); //Cuda sync
  
    //checking for any errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) 
        printf("Error: %s\n", hipGetErrorString(err)); 

    //copying the device array to CPU   
    hipMemcpy(array, xArrayPointer, n*sizeof(float), hipMemcpyDeviceToHost);
  
    end = std::chrono::system_clock::now();
    std::chrono::duration<double> totaltime = (end-begin)/nbiter;

    std::cerr<<array[0]<<std::endl;
    std::cout<<n<<" "<<degree<<" "<<totaltime.count()<<std::endl;

    delete[] array;
    delete[] poly;
    hipFree(xArrayPointer); //free cuda mem
    hipFree(coefficientArrayPointer);
    return 0;
}
